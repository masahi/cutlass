/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/epilogue/thread/scale_type.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_mma.hpp"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;
using namespace cutlass;
using namespace cutlass::gemm;

template <
  class StrideC_,
  class StrideD_,
  class ThreadEpilogueOp_
>
class EpilogueThreeSources {
public:
  using ThreadEpilogueOp = ThreadEpilogueOp_;
  using ElementOutput = typename ThreadEpilogueOp::ElementOutput;
  using ElementAccumulator = typename ThreadEpilogueOp::ElementAccumulator;
  using ElementCompute = typename ThreadEpilogueOp::ElementCompute;
  using ElementScalar = ElementCompute;
  using ElementC = typename ThreadEpilogueOp::ElementC;
  using StrideC = StrideC_;
  using ElementD = typename ThreadEpilogueOp::ElementD;
  using StrideD = StrideD_;

  static const int kOutputAlignment = ThreadEpilogueOp::kCount;
  using AlignmentType = typename cute::uint_bit<cutlass::sizeof_bits<ElementOutput>::value * kOutputAlignment>::type;

  static_assert(rank(StrideC{}) == 3, "StrideCD must be rank-3: [M, N, L]");
  static_assert(rank(StrideD{}) == 3, "StrideCD must be rank-3: [M, N, L]");

  struct SharedStorage { };

  struct Params {
    ElementC const* ptr_C1 = nullptr;
    ElementC const* ptr_C2 = nullptr;
    ElementC const* ptr_C3 = nullptr;
    StrideC dC{};
    ElementD* ptr_D = nullptr;
    StrideD dD{};
    typename ThreadEpilogueOp::Params thread_params{};
  };

  template <class Args>
  static constexpr Params
  to_underlying_arguments(Args const& args, void* workspace) {
    (void) workspace;
    return {args.epilogue_params};
  }

  CUTLASS_HOST_DEVICE
  EpilogueThreeSources(Params const& params_) : params(params_) { }

  template<
    class ProblemShapeMNKL,
    class BlockShapeMNK,
    class BlockCoordMNKL,
    class FrgEngine, class FrgLayout,
    class TiledMma,
    class ResidueMNK
  >
  CUTLASS_HOST_DEVICE void
  operator()(
      ProblemShapeMNKL problem_shape_mnkl,
      BlockShapeMNK blk_shape_MNK,
      BlockCoordMNKL blk_coord_mnkl,
      cute::Tensor<FrgEngine, FrgLayout> const& accumulators,
      TiledMma tiled_mma,
      ResidueMNK residue_mnk,
      int thread_idx,
      char* smem_buf)
  {
    using X = Underscore;

    static_assert(rank(ProblemShapeMNKL{}) == 4, "ProblemShapeMNKL must be rank 4");
    static_assert(is_static<BlockShapeMNK>::value, "ThreadBlock tile shape must be static");
    static_assert(rank(BlockShapeMNK{}) == 3, "BlockShapeMNK must be rank 3");
    static_assert(rank(BlockCoordMNKL{}) == 4, "BlockCoordMNKL must be rank 3");

    (void) smem_buf;
    ThreadEpilogueOp epilogue_op{params.thread_params};

    // Separate out problem shape for convenience
    auto M = get<0>(problem_shape_mnkl);
    auto N = get<1>(problem_shape_mnkl);
    auto L = get<3>(problem_shape_mnkl);
    // Slice to get the tile this CTA is responsible for
    auto [m_coord, n_coord, k_coord, l_coord] = blk_coord_mnkl;

    // Represent the full output tensor
    Tensor mD_mnl = make_tensor(make_gmem_ptr(params.ptr_D), make_shape(M,N,L), params.dD);                // (m,n,l)
    Tensor gD_mnl = local_tile(mD_mnl, blk_shape_MNK, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)
    Tensor gD = gD_mnl(_,_,m_coord,n_coord,l_coord);                                                 // (BLK_M,BLK_N)

    // Partition source and destination tiles to match the accumulator partitioning
    auto thr_mma = tiled_mma.get_thread_slice(thread_idx);
    Tensor tCgD = thr_mma.partition_C(gD);                                       // (VEC,THR_M,THR_N)

    // Make an identity coordinate tensor for predicating our output MN tile
    auto cD = make_identity_tensor(make_shape(unwrap(shape<0>(gD)), unwrap(shape<1>(gD))));
    Tensor tCcD = thr_mma.partition_C(cD);

    static_assert(is_static<FrgLayout>::value, "Accumulator layout must be static");
    CUTE_STATIC_ASSERT_V(size(tCgD) == size(accumulators),
        "Accumulator count must have the same destination element count.");

    Tensor mC1_mnl = make_tensor(make_gmem_ptr(params.ptr_C1), make_shape(M,N,L), params.dC);                // (m,n,l)
    Tensor gC1_mnl = local_tile(mC1_mnl, blk_shape_MNK, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)
    Tensor gC1 = gC1_mnl(_,_,m_coord,n_coord,l_coord);                                                 // (BLK_M,BLK_N)
    Tensor tCgC1 = thr_mma.partition_C(gC1);                                       // (VEC,THR_M,THR_N)

    Tensor mC2_mnl = make_tensor(make_gmem_ptr(params.ptr_C2), make_shape(M,N,L), params.dC);                // (m,n,l)
    Tensor gC2_mnl = local_tile(mC2_mnl, blk_shape_MNK, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)
    Tensor gC2 = gC2_mnl(_,_,m_coord,n_coord,l_coord);                                                 // (BLK_M,BLK_N)
    Tensor tCgC2 = thr_mma.partition_C(gC2);                                       // (VEC,THR_M,THR_N)

    Tensor mC3_mnl = make_tensor(make_gmem_ptr(params.ptr_C3), make_shape(M,N,L), params.dC);                // (m,n,l)
    Tensor gC3_mnl = local_tile(mC3_mnl, blk_shape_MNK, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)
    Tensor gC3 = gC3_mnl(_,_,m_coord,n_coord,l_coord);                                                 // (BLK_M,BLK_N)
    Tensor tCgC3 = thr_mma.partition_C(gC3);                                       // (VEC,THR_M,THR_N)

    CUTE_STATIC_ASSERT_V(size(tCgC1) == size(tCgD),
        "Source and destination must have the same number of elements.");

    CUTLASS_PRAGMA_UNROLL
    for (int i = 0; i < size(accumulators); ++i) {
      if (elem_less(tCcD(i), make_coord(get<0>(residue_mnk), get<1>(residue_mnk)))) {
        tCgD(i) = epilogue_op(accumulators(i), tCgC1(i) + tCgC2(i) + tCgC3(i));
      }
    }
  }

private:
  Params params;
};

// A matrix configuration
using         ElementA    = cutlass::half_t;                                          // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::half_t;                                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;                                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementOutput = cutlass::half_t;

using DispatchPolicy = MainloopSm80CpAsync<3>;

using TiledMma = TiledMMA<
      MMA_Atom<SM80_16x8x16_F32F16F16F32_TN>,	Layout<Shape<_2,_2,_1>>,  // 2x2x1 thread group
	Layout<Shape<_1,_2,_1>>>; // 1x2x1 value group for 16x16x16 MMA and LDSM

template <typename Element, typename Layout, int Alignment, int SizeK>
struct DefaultGemm_TensorOpSm80_OperandA;

template <typename Element, typename Layout, int Alignment, int SizeK>
struct DefaultGemm_TensorOpSm80_OperandB;

template <>
struct DefaultGemm_TensorOpSm80_OperandA<half_t, layout::RowMajor, 8, 64>
{
  // Smem
  using SmemLayoutAtom = decltype(
				  composition(Swizzle<3,3,3>{},
					      Layout<Shape < _8,_64>,
					      Stride<_64, _1>>{}));
  using SmemCopyAtom = Copy_Atom<SM75_U32x4_LDSM_N, half_t>;

  // Gmem
  using GmemTiledCopy = decltype(
				 make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, half_t>{},
						 Layout<Shape <_16,_8>,
						 Stride< _8,_1>>{},
						 Layout<Shape < _1,_8>>{}));
};

// Because the F32F16 TiledMMA is A-B symmetric, we can reuse the DefaultOperands

// Operand B - Column-Major (K-major)
template <int Alignment, int SizeK>
struct DefaultGemm_TensorOpSm80_OperandB<half_t, layout::ColumnMajor, Alignment, SizeK>
  : DefaultGemm_TensorOpSm80_OperandA<half_t, layout::RowMajor,    Alignment, SizeK>
{};

// Operand B - Row-Major (N-major)
template <int Alignment, int SizeK>
struct DefaultGemm_TensorOpSm80_OperandB<half_t, layout::RowMajor,    Alignment, SizeK>
  : DefaultGemm_TensorOpSm80_OperandA<half_t, layout::ColumnMajor, Alignment, SizeK>
{};

//
// F16: 128-by-128-by-32 (small k-block)
//

/// Operand A - Row-major (K-Major)
template <>
struct DefaultGemm_TensorOpSm80_OperandA<half_t, layout::RowMajor, 8, 32>
{
  // Smem
  using SmemLayoutAtom = decltype(
				  composition(Swizzle<2,3,3>{},
					      Layout<Shape < _8,_32>,
					      Stride<_32, _1>>{}));
  using SmemCopyAtom = Copy_Atom<SM75_U32x4_LDSM_N, half_t>;

  // Gmem
  using GmemTiledCopy = decltype(
				 make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, half_t>{},
						 Layout<Shape <_32,_4>,
						 Stride< _4,_1>>{},
						 Layout<Shape < _1,_8>>{}));
};

using DefaultOperandA = DefaultGemm_TensorOpSm80_OperandA<
    half_t, LayoutA, AlignmentA, 32>;

using SmemLayoutAtomA = DefaultOperandA::SmemLayoutAtom; // M, K
using SmemCopyAtomA = DefaultOperandA::SmemCopyAtom;
using GmemTiledCopyA = DefaultOperandA::GmemTiledCopy;

// B
using DefaultOperandB = DefaultGemm_TensorOpSm80_OperandB<
    half_t, LayoutB, AlignmentB, 32>;
using SmemLayoutAtomB =  DefaultOperandB::SmemLayoutAtom; // N, K
using SmemCopyAtomB =  DefaultOperandB::SmemCopyAtom;
using GmemTiledCopyB =  DefaultOperandB::GmemTiledCopy;

using CollectiveEpilogue = EpilogueThreeSources<
    cutlass::gemm::TagToStrideC_t<LayoutC>,
      cutlass::gemm::TagToStrideC_t<LayoutC>,
      cutlass::epilogue::thread::LinearCombination<ElementC, 1, ElementAccumulator, ElementAccumulator,
						     epilogue::thread::ScaleType::Default,
						     FloatRoundStyle::round_to_nearest,
						     ElementOutput>>;

using TileShape          = Shape<_128,_128,_32>;                           // Threadblock-level tile size

using CollectiveMainloop = collective::CollectiveMma<
    DispatchPolicy, TileShape,
      half_t, TagToStrideA_t<LayoutA>,
      half_t, TagToStrideB_t<LayoutB>,
      TiledMma,
      GmemTiledCopyA, SmemLayoutAtomA, SmemCopyAtomA, cute::identity,  // A
      GmemTiledCopyB, SmemLayoutAtomB, SmemCopyAtomB, cute::identity   // B
      >;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int>, // Indicates ProblemShape
      CollectiveMainloop,
      CollectiveEpilogue
      >;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
    LayoutA,
    ElementB,
    LayoutB,
    ElementC,
    LayoutC,
    ElementAccumulator,
    ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C1;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C2;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C3;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;

  Options():
    help(false),
    m(5120), n(4096), k(4096),
    alpha(1.f), beta(0.f),
    iterations(1000)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "48_hopper_warp_specialized_gemm\n\n"
	<< "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
	<< "Options:\n\n"
	<< "  --help                      If specified, displays this usage statement\n\n"
	<< "  --m=<int>                   Sets the M extent of the GEMM\n"
	<< "  --n=<int>                   Sets the N extent of the GEMM\n"
	<< "  --k=<int>                   Sets the K extent of the GEMM\n"
	<< "  --alpha=<f32>               Epilogue scalar alpha\n"
	<< "  --beta=<f32>                Epilogue scalar beta\n\n"
	<< "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "48_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
	 double avg_runtime_ms = 0,
	 double gflops = 0,
	 cutlass::Status status = cutlass::Status::kSuccess,
	 hipError_t error = hipSuccess)
    :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
		      cutlass::DeviceAllocation<Element>& block,
		      uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
						     block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, Int<1>{}));
  stride_B = make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, Int<1>{}));
  stride_C = make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, Int<1>{}));
  stride_D = make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, Int<1>{}));

  block_A.reset(options.m * options.k);
  block_B.reset(options.k * options.n);
  block_C1.reset(options.m * options.n);
  block_C2.reset(options.m * options.n);
  block_C3.reset(options.m * options.n);
  block_D.reset(options.m * options.n);
  block_ref_D.reset(options.m * options.n);

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C1, seed + 2021);
  initialize_block(block_C2, seed + 2021);
  initialize_block(block_C3, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
      {options.m, options.n, options.k},
      block_A.get(),
      stride_A,
      block_B.get(),
      stride_B,
      {block_C1.get(), block_C2.get(), block_C3.get(), stride_C, block_D.get(), stride_D, {options.alpha, options.beta}}
  };

  return arguments;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  Options options;
  options.parse(argc, args);

  run<Gemm>(options);
  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
